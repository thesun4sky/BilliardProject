#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "TFManager.h"

#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_vector_types.h>

const float PI = 3.1415926536f;
typedef unsigned short ushort;
typedef unsigned char uchar;

texture<ushort, 3, hipReadModeNormalizedFloat> tex_volume;  // 3D texture
texture<ushort, 3, hipReadModeNormalizedFloat> tex_block; //Block texture
texture<float4, 1, hipReadModeElementType> tex_TF;
texture<float4, 3, hipReadModeElementType> tex_TF2d;		//pre-integral

texture<float, 3, hipReadModeElementType> tex_average; //Average texture
texture<float, 3, hipReadModeElementType> tex_sigma;	//Sigma texture
texture<float, 3, hipReadModeElementType> tex_average_half; //Average texture
texture<float, 3, hipReadModeElementType> tex_sigma_half;	//Sigma texture

hipArray *d_volumeArray = 0;
hipArray *d_blockArray = 0;
hipArray *d_TFArray = 0;
hipArray *d_TF2dArray = 0;
hipArray *d_AverageArray = 0;
hipArray *d_SigmaArray = 0;
hipArray *d_AverageHalfArray = 0;
hipArray *d_SigmaHalfArray = 0;

extern "C"
{
	void FreeGPUVolArray(void)
	{
		hipFreeArray(d_volumeArray);
		hipFreeArray(d_blockArray);
	}

	void FreeGPUTFArray(void)
	{
		hipFreeArray(d_TFArray);
	}

	void FreeGPUEtcArray(void)
	{
		hipFreeArray(d_AverageArray);
		hipFreeArray(d_SigmaArray);
	}
}


void initTFTexture(int width, float4 *h_data)   
{
	if(d_TFArray != 0)
		hipFreeArray(d_TFArray);

	uint size = width*sizeof(float)*4;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	checkCudaErrors(hipMallocArray(&d_TFArray, &channelDesc, width)); 
	
	checkCudaErrors(hipMemcpyToArray(d_TFArray, 0, 0, h_data, size, hipMemcpyHostToDevice));

    tex_TF.addressMode[0] = hipAddressModeClamp;
    tex_TF.addressMode[1] = hipAddressModeClamp;
    tex_TF.filterMode = hipFilterModePoint;
    tex_TF.normalized = false;    // access with integer texture coordinates
	checkCudaErrors(hipBindTextureToArray(tex_TF, d_TFArray, channelDesc));

}


void initVolume(const ushort *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
	if(d_volumeArray != NULL) {
		hipFreeArray(d_volumeArray);
		d_volumeArray=NULL;
	}
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize, 0) );

    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_volumeArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_volume.normalized = false;                      // access with normalized texture coordinates
    tex_volume.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_volume.channelDesc = channelDesc;
	tex_volume.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_volume.addressMode[1] = hipAddressModeBorder;
    tex_volume.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_volume, d_volumeArray, channelDesc));
}


void initAvgVolume(const float *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_AverageArray, &channelDesc, volumeSize, 0) );

    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_AverageArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_average.normalized = false;                      // access with normalized texture coordinates
    tex_average.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_average.channelDesc = channelDesc;
	tex_average.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_average.addressMode[1] = hipAddressModeBorder;
    tex_average.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_average, d_AverageArray, channelDesc));
}

void initAvgHalfVolume(const float *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_AverageHalfArray, &channelDesc, volumeSize, 0) );

    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_AverageHalfArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_average_half.normalized = false;                      // access with normalized texture coordinates
    tex_average_half.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_average_half.channelDesc = channelDesc;
	tex_average_half.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_average_half.addressMode[1] = hipAddressModeBorder;
    tex_average_half.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_average_half, d_AverageHalfArray, channelDesc));
}


void initSigVolume(const float *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_SigmaArray, &channelDesc, volumeSize, 0) );

    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_SigmaArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_sigma.normalized = false;                      // access with normalized texture coordinates
    tex_sigma.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_sigma.channelDesc = channelDesc;
	tex_sigma.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_sigma.addressMode[1] = hipAddressModeBorder;
    tex_sigma.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_sigma, d_SigmaArray, channelDesc));
}

void initSigHalfVolume(const float *h_volume, hipExtent volumeSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_SigmaHalfArray, &channelDesc, volumeSize, 0) );

    // copy data to 3D array
	int x = volumeSize.width;
	int y = volumeSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_SigmaHalfArray;
    myParams.extent   = volumeSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_sigma_half.normalized = false;                      // access with normalized texture coordinates
    tex_sigma_half.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_sigma_half.channelDesc = channelDesc;
	tex_sigma_half.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_sigma_half.addressMode[1] = hipAddressModeBorder;
    tex_sigma_half.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_sigma_half, d_SigmaHalfArray, channelDesc));
}


void initBlockTexture(const ushort *h_volume_block, hipExtent blockSize, int bytePerVoxel)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors( hipMalloc3DArray(&d_blockArray, &channelDesc, blockSize, 0) );

    // copy data to 3D array
	int x = blockSize.width;
	int y = blockSize.height;
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume_block, x*bytePerVoxel, x, y);
    myParams.dstArray = d_blockArray;
    myParams.extent   = blockSize;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_block.normalized = false;                      // access with normalized texture coordinates
    tex_block.filterMode = hipFilterModePoint;      // linear interpolation
    tex_block.channelDesc = channelDesc;
	tex_block.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_block.addressMode[1] = hipAddressModeBorder;
    tex_block.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_block, d_blockArray, channelDesc));            
} 



__device__ void GetRayBound(float *t, float3 sdot, float3 start, hipExtent volumeSize){

	const float EPS = 0.00001; // epsilon
	// [0,0,0] ~ [255,255,224] box
	// eye : sdot
	// direction : start
	// get t1, t2
	float kx[2]={-20000,20000}, ky[2]={-20000,20000}, kz[2]={-20000,20000};
	// sdot.x + kx[0] * start.x = 0;
	if( fabs((float)start.x) > EPS) {
		kx[0] = (0 - sdot.x) / start.x;
		kx[1] = (volumeSize.width - sdot.x) / start.x;
		if( kx[0] > kx[1] ) { // in > out
			float temp = kx[0];
			kx[0] = kx[1];
			kx[1] = temp;
		}
	}

	if( fabs((float)start.y) > EPS){
		ky[0] = (0 - sdot.y) / start.y;
		ky[1] = (volumeSize.height - sdot.y) / start.y;
		if( ky[0] > ky[1] ) { // in > out
			float temp = ky[0];
			ky[0] = ky[1];
			ky[1] = temp;
		}
	}

	if( fabs((float)start.z) > EPS){
		kz[0] = (0 - sdot.z) / start.z;
		kz[1] = (volumeSize.depth - sdot.z) / start.z;
		if( kz[0] > kz[1] ) { // in > out
			float temp = kz[0];
			kz[0] = kz[1];
			kz[1] = temp;
		}
	}

	float kin = max(max(kx[0], ky[0]), kz[0]);
	float kout = min(min(kx[1], ky[1]), kz[1]);

	t[0] = kin + 0.05f;
	t[1] = kout - 0.05f;

}

__device__ ushort myMAX(ushort a, ushort b)
{
	if(a >= b)
		return a;
	else 
		return b;
}


__device__ float GetSum(float Average, float Sigma, int start, int end, float* probability_k)
{
	//start=start*16.0f;
	//end=end*16.0f;
	//Average = Average-200.0f;
	float startz = (start - Average)/Sigma;
	float endz = (end - Average)/Sigma;
	float pi = 3.141592f, e = 2.718f;
	float p1=0.0f, p2=0.0f;

	if(startz > 5.0f)
		p1 = 1.0f;
	else if(startz < -5.0f)
		p1 = 0.0f;
	//else if(startz < -10.0f)
	//	p1 = 0.5f;
	else if(startz >= 0.0f)
		p1 = 0.5f + probability_k[(int)(startz*100)];
	else 
		p1 = 0.5f - probability_k[-(int)(startz*100)];

	if(endz > 5.0f)
		p2 = 1.0f;
	else if(endz < -5.0f)
		p2 = 0.0f;
	//else if(endz < -10.0f)
	//	p2 = 0.5f;
	else if(endz >= 0.0f)
		p2 = 0.5f + probability_k[(int)(endz*100)];
	else 
		p2 = 0.5f - probability_k[-(int)(endz*100)];

	if(endz == startz)
		endz = startz+0.1f; //debug code

	
	return ((1.0f/((endz-startz)*sqrt(2.0f*pi)))*((1.0f/pow(sqrt(e), startz*startz))-
		(1.0f/pow(sqrt(e), endz*endz))) + (-startz*1.0f/(endz-startz))*(p2-p1) + 1.0f*(1-p2));
}


//a,b,c,d �� ���� �̿��� AO SUM �Լ�.
//���� ��ȯ�Լ��� ��ٸ��÷� �ٲٸ� �ȴ�. (2015.04.27)
__device__ float GetSum(float Average, float Sigma, int a, int b, int c, int d, float* probability_k)
{
	float pi = 3.141592f, e = 2.718f;

	float starta = (a - Average)/Sigma;
	float endb = (b - Average)/Sigma;
	float p1=0.0f, p2=0.0f;

	if(starta > 5.0f)
		p1 = 1.0f;
	else if(starta < -5.0f)
		p1 = 0.0f;
	//else if(startz < -10.0f)
	//	p1 = 0.5f;
	else if(starta >= 0.0f)
		p1 = 0.5f + probability_k[(int)(starta*100)];
	else 
		p1 = 0.5f - probability_k[-(int)(starta*100)];

	if(endb > 5.0f)
		p2 = 1.0f;
	else if(endb < -5.0f)
		p2 = 0.0f;
	else if(endb >= 0.0f)
		p2 = 0.5f + probability_k[(int)(endb*100)];
	else 
		p2 = 0.5f - probability_k[-(int)(endb*100)];

	float startc = (c - Average)/Sigma;
	float endd = (d - Average)/Sigma;
	float p3=0.0f, p4=0.0f;

	if(startc > 5.0f)
		p3 = 1.0f;
	else if(startc < -5.0f)
		p3 = 0.0f;
	else if(startc >= 0.0f)
		p3 = 0.5f + probability_k[(int)(startc*100)];
	else 
		p3 = 0.5f - probability_k[-(int)(startc*100)];

	if(endd > 5.0f)
		p4 = 1.0f;
	else if(endd < -5.0f)
		p4 = 0.0f;
	else if(endd >= 0.0f)
		p4 = 0.5f + probability_k[(int)(endd*100)];
	else 
		p4 = 0.5f - probability_k[-(int)(endd*100)];
	

	//return ((1.0f/((endz-startz)*sqrt(2.0f*pi)))*((1.0f/pow(sqrt(e), startz*startz))-
	//		 (1.0f/pow(sqrt(e), endz*endz))) + (-startz*1.0f/(endz-startz))*(p2-p1) + 1.0f*(1-p2));

	return (1.0f/((endb-starta)*sqrtf(2.0f*pi)))*((1.0f/pow(sqrtf(e), starta*starta))-
			(1.0f/pow(sqrtf(e), endb*endb))) + (-starta/(endb-starta))*(p2-p1)
			+ (p3-p2) 
				+ (1.0f/((startc-endd)*sqrtf(2.0f*pi)))*((1.0f/pow(sqrtf(e), startc*startc))-
				(1.0f/pow(sqrtf(e), endd*endd))) + (-endd/(startc-endd))*(p4-p3);
}


__global__ void makeBlock_kernel(ushort* image_p, ushort* dest_p, hipExtent blockSize, hipExtent volumeSize)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (tx >= blockSize.width || ty >= blockSize.height) return;

	for(int i=0; i<blockSize.depth; i++){
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = 0;
		ushort tempmax=0;

		for(int z=i*4; z<=i*4+4; z++)
			for(int y=ty*4; y<=ty*4+4; y++)
				for(int x=tx*4; x<=tx*4+4; x++){
					if(z>=volumeSize.depth || y>=volumeSize.height || x>=volumeSize.width )
						continue;
					tempmax = myMAX(tempmax, image_p[z*volumeSize.width*volumeSize.height + y*volumeSize.height + x]);
				}
		dest_p[i*blockSize.width*blockSize.height + ty*blockSize.height + tx] = tempmax;
	}

		
}


__global__ void cuda_kernel(uchar *surface, int width, int height, hipExtent volumeSize, float3 sdot, 
							float3 vDir, float3 vXcross, float3 vYcross, float zResolution, float blockResolution)
{
    int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
/*
    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (tx >= width || ty >= height) return;

	sdot = sdot + (tx-width/2)*vXcross + (ty-height/2)*vYcross;

	float t[2] = {0.0f, 1000.0f};
	GetRayBound(t, sdot, vDir, volumeSize); //t1, t2�޾ƿ���

	float4 intensity = {0.0f};
	float alpha = 0.0f;
	bool bShading=false;
	bool bSkipping = false;

	for(float i=t[0]; i<t[1]; i+=1.0f){

		float3 render={0.0f, 0.0f, 0.0f};
		render = sdot + i*vDir;

		float block_den = tex3D(tex_block, (int)(render.x*blockResolution), (int)(render.y*blockResolution), 
								int(render.z*blockResolution))*65535;
		float3 advanced  = {0.0f, 0.0f, 0.0f};
		if((int)block_den < alpha_start) { 
			int3 nowPos = {(int)(render.x*blockResolution), (int)(render.y*blockResolution), 
							(int)(render.z*blockResolution)};
			int3 advPos;
			do {
				i += 1.0f;
				advanced = sdot + i*vDir;
				advPos.x = (int)(advanced.x*blockResolution);
				advPos.y = (int)(advanced.y*blockResolution);
				advPos.z = (int)(advanced.z*blockResolution);

			} while ( nowPos.x == advPos.x &&
					  nowPos.y == advPos.y &&
					  nowPos.z == advPos.z);
			i -= 1.0f;
			bShading=true;
			bSkipping=true;
			continue;
		}

		float den = tex3D(tex_volume, render.x, render.y, render.z)*65535;
		//float den_next = tex3D(tex_volume, render.x+startvec.x, render.y+startvec.y, render.z+startvec.z)*4095; //next voxel
	
		float4 samplecolor = tex1D(tex_TF, den);
		//float4 samplecolor = tex3D(tex_TF2d, den, den_next, 0); //pre-integral 

		if(bSkipping){
			float3 prevpos = sdot +(i-1)*vDir;
			float den_prev = tex3D(tex_volume, prevpos.x, prevpos.y, prevpos.z)*65535;
			float4 prevcolor = tex1D(tex_TF, den_prev);
		
			samplecolor +=  (1.0f-samplecolor.w)*prevcolor*prevcolor.w;
		}
		bSkipping=false;

		if(samplecolor.w < 0.01f) {} else
		if(samplecolor.w > 0.001f && bShading){
			//------------------------------------------------------------------------
			//shading1 - local - NL�� �̾Ƴ���
			//float shading1 = 0.0f;
			float3 nV = {0.0, 0.0, 0.0};
			float3 lV = {0.0, 0.0, 0.0};

			lV = vDir;

			float x_plus = tex3D(tex_volume, render.x+1, render.y, render.z)*65535;
			float x_minus = tex3D(tex_volume, render.x-1, render.y, render.z)*65535;

			float y_plus = tex3D(tex_volume, render.x, render.y+1, render.z)*65535;
			float y_minus = tex3D(tex_volume, render.x, render.y-1, render.z)*65535;

			float z_plus = tex3D(tex_volume, render.x, render.y, render.z+1)*65535;
			float z_minus = tex3D(tex_volume, render.x, render.y, render.z-1)*65535;

			nV.x = (x_plus - x_minus);
			nV.y = (y_plus - y_minus);
			nV.z = (z_plus - z_minus)*(float)zResolution;

			nV = normalize(nV);

			float NL = 0.0f;
			NL = lV.x*nV.x + lV.y*nV.y + lV.z*nV.z;

			if(NL < 0.0f) NL = 0.0f;

			float localShading = 0.3 + 0.7*NL;

			samplecolor.x *= localShading;
			samplecolor.y *= localShading;
			samplecolor.z *= localShading;
		} else
		{
			const float fCutPlaneShading = 0.0f;
			samplecolor = samplecolor*fCutPlaneShading;
		
		}
		bShading = true;

		intensity.x += (1.0f-alpha)*samplecolor.x*samplecolor.w;
		intensity.y += (1.0f-alpha)*samplecolor.y*samplecolor.w;
		intensity.z += (1.0f-alpha)*samplecolor.z*samplecolor.w;
		alpha += (1.0f-alpha)*samplecolor.w;

		if(alpha > 0.95f)
			break;

	}

	surface[(ty*width + tx)*3 + 0] = intensity.x;
	surface[(ty*width + tx)*3 + 1] = intensity.y;
	surface[(ty*width + tx)*3 + 2] = intensity.z;
	*/
	surface[(ty*width + tx)*3 + 0] = 255;
	surface[(ty*width + tx)*3 + 1] = 0;
	surface[(ty*width + tx)*3 + 2] = 0;

}

__global__ void cuda_kernel_AO(uchar *surface, int width, int height, hipExtent volumeSize, float3 sdot, 
							float3 vDir, float3 vXcross, float3 vYcross, float zResolution, float blockResolution,
							float* probability_k, float3 factor)
{
    int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (tx >= width || ty >= height) return;

	sdot = sdot + (tx-width/2)*vXcross + (ty-height/2)*vYcross;

	float t[2] = {0.0f, 1000.0f};
	GetRayBound(t, sdot, vDir, volumeSize); //t1, t2�޾ƿ���

	float4 intensity = {0.0f};
	float alpha = 0.0f;
	bool bShading=false;
	bool bSkipping = false;

	for(float i=t[0]; i<t[1]; i+=1.0f){

		float3 render={0.0f, 0.0f, 0.0f};
		render = sdot + i*vDir;

		float block_den = tex3D(tex_block, (int)(render.x*blockResolution), (int)(render.y*blockResolution), 
								int(render.z*blockResolution))*65535;
		float3 advanced  = {0.0f, 0.0f, 0.0f};
		if((int)block_den < alpha_start) { 
			int3 nowPos = {(int)(render.x*blockResolution), (int)(render.y*blockResolution), 
							(int)(render.z*blockResolution)};
			int3 advPos;
			do {
				i += 1.0f;
				advanced = sdot + i*vDir;
				advPos.x = (int)(advanced.x*blockResolution);
				advPos.y = (int)(advanced.y*blockResolution);
				advPos.z = (int)(advanced.z*blockResolution);

			} while ( nowPos.x == advPos.x &&
					  nowPos.y == advPos.y &&
					  nowPos.z == advPos.z);
			i -= 1.0f;
			bShading=true;
			bSkipping=true;
			continue;
		}

		float den = tex3D(tex_volume, render.x, render.y, render.z)*65535;
		//float den_next = tex3D(tex_volume, render.x+startvec.x, render.y+startvec.y, render.z+startvec.z)*4095; //next voxel
	
		float4 samplecolor = tex1D(tex_TF, den);
		//float4 samplecolor = tex3D(tex_TF2d, den, den_next, 0); //pre-integral 

		if(bSkipping){
			float3 prevpos = sdot +(i-1)*vDir;
			float den_prev = tex3D(tex_volume, prevpos.x, prevpos.y, prevpos.z)*65535;
			float4 prevcolor = tex1D(tex_TF, den_prev);
		
			samplecolor +=  (1.0f-samplecolor.w)*prevcolor*prevcolor.w;
		}
		bSkipping=false;

		if(samplecolor.w < 0.01f) {} else
		if(samplecolor.w > 0.001f && bShading){
			//------------------------------------------------------------------------
			//shading1 - local - NL�� �̾Ƴ���
			//float shading1 = 0.0f;
			float3 nV = {0.0, 0.0, 0.0};
			float3 lV = {0.0, 0.0, 0.0};

			lV = vDir;

			float x_plus = tex3D(tex_volume, render.x+1, render.y, render.z)*65535;
			float x_minus = tex3D(tex_volume, render.x-1, render.y, render.z)*65535;

			float y_plus = tex3D(tex_volume, render.x, render.y+1, render.z)*65535;
			float y_minus = tex3D(tex_volume, render.x, render.y-1, render.z)*65535;

			float z_plus = tex3D(tex_volume, render.x, render.y, render.z+1)*65535;
			float z_minus = tex3D(tex_volume, render.x, render.y, render.z-1)*65535;

			nV.x = (x_plus - x_minus);
			nV.y = (y_plus - y_minus);
			nV.z = (z_plus - z_minus)/zResolution;

			nV = normalize(nV);

			float NL = lV.x*nV.x + lV.y*nV.y + lV.z*nV.z;
			NL = max(0.f, NL);

			float localShading = 0.3 + 0.7*NL;
			//------------------------------------------------------------------------
			//shading2 - global
			float3 modify_render = render*0.5f;
			float Sigma = tex3D(tex_sigma, modify_render.x, modify_render.y, modify_render.z);
			float Average = tex3D(tex_average, modify_render.x, modify_render.y, modify_render.z);	

			//nV *= 1.f;
			//cubesize 7 sigma, average
			//float Sigma_half = tex3D(tex_sigma_half, modify_render.x+nV.x, modify_render.y+nV.y, modify_render.z+nV.z);
			//float Average_half = tex3D(tex_average_half, modify_render.x+nV.x, modify_render.y+nV.y, modify_render.z+nV.z);	

			//samplecolor = tex1D(tex_TF, Average);

			float sum = GetSum(Average, Sigma, alpha_start, alpha_end, probability_k); 
			//float sum_half = GetSum(Average_half, Sigma_half, alpha_start, alpha_end, probability_k);

			//float size0 = 7.f;
			//float size1 = 5.f;
			//float fVal[2] = {size0*size0*size0, size1*size1*size1};
			//float new_sum = (fVal[0]*sum - fVal[1]*sum_half)/(fVal[0]-fVal[1]);
				
			float shading2 = 1.0f - min(max((sum*2.0f - 0.5f), 0.0f), 1.0f); //global shding value ����

			float shading = factor.x + factor.y*shading2*shading2 + factor.z*NL; //factor1,2,3

			samplecolor.x *= shading2;
			samplecolor.y *= shading2;
			samplecolor.z *= shading2;
		} else
		{
			const float fCutPlaneShading = 0.0f;
			samplecolor = samplecolor*fCutPlaneShading;
		
		}
		bShading = true;

		intensity.x += (1.0f-alpha)*samplecolor.x*samplecolor.w;
		intensity.y += (1.0f-alpha)*samplecolor.y*samplecolor.w;
		intensity.z += (1.0f-alpha)*samplecolor.z*samplecolor.w;
		alpha += (1.0f-alpha)*samplecolor.w;

		if(alpha > 0.95f)
			break;

	}

	surface[(ty*width + tx)*3 + 0] = intensity.x;
	surface[(ty*width + tx)*3 + 1] = intensity.y;
	surface[(ty*width + tx)*3 + 2] = intensity.z;
}

ushort* make_blockVolume(ushort* image, hipExtent blockSize, hipExtent volumeSize)
{
	unsigned int vsize = volumeSize.width * volumeSize.height * volumeSize.depth * sizeof(ushort);
	unsigned int bsize = blockSize.width * blockSize.height * blockSize.depth * sizeof(ushort);

	ushort *dest, *dest_p, *image_p;

	dest = new ushort[bsize/sizeof(ushort)];
	memset((void*)dest, 0, bsize);

	hipMalloc((void**)&image_p, vsize);
	hipMemcpy(image_p, image, vsize, hipMemcpyHostToDevice);

	hipMalloc((void**)&dest_p, bsize);

	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((blockSize.width+Db.x-1)/Db.x, (blockSize.height+Db.y-1)/Db.y);

	makeBlock_kernel<<<Dg, Db>>>(image_p, dest_p, blockSize, volumeSize);

	hipMemcpy(dest, dest_p, bsize, hipMemcpyDeviceToHost);

	hipFree(image_p);
	hipFree(dest_p);

	return dest;

}


void Run_Kernel(uchar* surface, const int imgsize[2], hipExtent volumeSize, ushort* pVol,
				float zResolution, float blockResolution, const float *ViewingPoint)
{
	printf("-GPU render : Basic\n");
	//---------------------------------------------------------------
	//����, ī�޶� ���� ����
	float3 volCenter = {volumeSize.width/2.0f, volumeSize.height/2.0f, volumeSize.depth/2.0f};
	float3 sdot={ViewingPoint[0], ViewingPoint[1], ViewingPoint[2]}, vUp={0.0f, 0.0f, 1.0f};
	float3 frontView = {volumeSize.width/2.f, volumeSize.height, volumeSize.depth/2.f};
	
	float3 vDir, vXCross, vYcross, front;
 
	front = frontView-volCenter;
	front = normalize(front);

	vDir = volCenter-sdot;
	vDir = normalize(vDir);

	float3 temp_z = {0.f, vDir.y, vDir.z};
	temp_z = normalize(temp_z);
	if(dot(front, temp_z) < 0.f)
		vUp.z = -1.0f;
	
	vXCross = cross(vUp, vDir);
	vXCross = normalize(vXCross);
 
	vYcross = cross(vDir, vXCross);
	vYcross = normalize(vYcross);
	//---------------------------------------------------------------
	hipError_t ret;
	uchar* surface_k=NULL;
	ret = hipMalloc((void**)&surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	ret = hipMemset(surface_k, 0, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	ret = hipMemcpy(surface_k, surface, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyHostToDevice);

    //dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((imgsize[0]+Db.x-1)/Db.x, (imgsize[1]+Db.y-1)/Db.y);

    cuda_kernel<<<Dg,Db>>>(surface_k, imgsize[0], imgsize[1], volumeSize, sdot, vDir, 
		vXCross, vYcross, zResolution, blockResolution);
    if (hipGetLastError() != hipSuccess)
        printf("cuda_kernel() failed to launch error = %d\n", hipGetLastError());    

	ret = hipMemcpy(surface, surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyDeviceToHost);
	ret = hipFree(surface_k);
}


void Run_Kernel_AO(uchar* surface, const int imgsize[2], hipExtent volumeSize, ushort* pVol,
				float zResolution, float blockResolution, float probability[310], float factor[3], const float *ViewingPoint)
{
	printf("-GPU render : AO \n");
	//---------------------------------------------------------------
	//����, ī�޶� ���� ����
	float3 volCenter = {volumeSize.width/2.0f, volumeSize.height/2.0f, volumeSize.depth/2.0f};
	float3 sdot={ViewingPoint[0], ViewingPoint[1], ViewingPoint[2]}, vUp={0.0f, 0.0f, 1.0f};
	float3 frontView = {volumeSize.width/2.f, volumeSize.height, volumeSize.depth/2.f};
	
	float3 vDir, vXCross, vYcross, front;
 
	front = frontView-volCenter;
	front = normalize(front);

	vDir = volCenter-sdot;
	vDir = normalize(vDir);

	float3 temp_z = {0.f, vDir.y, vDir.z};
	temp_z = normalize(temp_z);
	if(dot(front, temp_z) < 0.f)
		vUp.z = -1.0f;
	
	vXCross = cross(vUp, vDir);
	vXCross = normalize(vXCross);
 
	vYcross = cross(vDir, vXCross);
	vYcross = normalize(vYcross);
	//---------------------------------------------------------------

	float* probability_k;
	hipMalloc((void**)&probability_k, 310*sizeof(float));
	hipMemset(probability_k, 0, 310*sizeof(float));
	hipMemcpy(probability_k, probability, 310*sizeof(float), hipMemcpyHostToDevice);
	
	float3 factor3 ={factor[0], factor[1], factor[2]};

	uchar* surface_k;
	hipMalloc((void**)&surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	hipMemset(surface_k, 0, imgsize[0]*imgsize[1]*3*sizeof(uchar));
	hipMemcpy(surface_k, surface, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyHostToDevice);

    //dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((imgsize[0]+Db.x-1)/Db.x, (imgsize[1]+Db.y-1)/Db.y);

    cuda_kernel_AO<<<Dg,Db>>>(surface_k, imgsize[0], imgsize[1], volumeSize, sdot, vDir, 
		vXCross, vYcross, zResolution, blockResolution, probability_k, factor3);

    if (hipGetLastError() != hipSuccess)
        printf("cuda_kernel() failed to launch error = %d\n", hipGetLastError());
    
	hipMemcpy(surface, surface_k, imgsize[0]*imgsize[1]*3*sizeof(uchar), hipMemcpyDeviceToHost);
	hipFree(surface_k);
}


void initTF2dTexture(float4 *h_volume, int x, int y, int z)
{
	hipExtent Size = make_hipExtent(x, y, z);
    // create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    checkCudaErrors( hipMalloc3DArray(&d_TF2dArray, &channelDesc, Size, 0) );

    // copy data to 3D array
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*sizeof(float4), x, y);
    myParams.dstArray = d_TF2dArray;
    myParams.extent   = Size;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_TF2d.normalized = false;                      // access with normalized texture coordinates
    tex_TF2d.filterMode = hipFilterModePoint;      // linear interpolation
    tex_TF2d.channelDesc = channelDesc;
	tex_TF2d.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_TF2d.addressMode[1] = hipAddressModeBorder;
    tex_TF2d.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_TF2d, d_TF2dArray, channelDesc));
}



__global__ void TF2d_kernel(float4* TF2d_k, int TFSize)
{
	int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	if(x>=TFSize || y>=TFSize)
		return;

	//float4 result;				//1�� ��� - pre-integral : OTF �����ϰ� �ص� �Ѱ㸸 ������ �Ҽ��ִ�.
	//float4 temp = {0.0f};
	//
	//if(y > x){
	//	for(int i=x; i<y; i++){
	//		temp = tex1D(tex_TF, i);

	//		float diff = i-x;

	//		if(diff == 0.0f)
	//			diff = 1.0f;

	//		temp.w = 1.0f-pow(1-temp.w, 1/diff);

	//		result.x += (1-result.w)*temp.x*temp.w;
	//		result.y += (1-result.w)*temp.y*temp.w;
	//		result.z += (1-result.w)*temp.z*temp.w;
	//		result.w += (1-result.w)*temp.w;
	//	}
	//}
	//else if(x > y){
	//	for(int i=y; i<x; i++){
	//		temp = tex1D(tex_TF, i);

	//		float diff = i-y;

	//		if(diff == 0.0f)
	//			diff = 1.0f;

	//		temp.w = 1.0f-pow(1-temp.w, 1/diff);

	//		result.x += (1-result.w)*temp.x*temp.w;
	//		result.y += (1-result.w)*temp.y*temp.w;
	//		result.z += (1-result.w)*temp.z*temp.w;
	//		result.w += (1-result.w)*temp.w;
	//	}
	//}
	//else {
	//	result.x = 255.0f;
	//	result.y = 255.0f;
	//	result.z = 255.0f;
	//	result.w = 0.0f;
	//}

	float4 temp;					//2�� ��� - 1��������� ���ṫ�̰� �� ����� : summed 2d table
	float4 result = {0.0};
	float4 sum = {0.0f};
	
	int nx, ny, diff;
	if(x>y){
		diff = x-y;
		ny = x;
		nx = y;
	}
	else if(y>x){
		diff = y-x;
		nx = x;
		ny = y;
	}
	else{
		diff=1;
		nx = ny = x;
		sum.w = 0.0f;
	}

	for(int i=nx; i<ny; i++){
		temp = tex1D(tex_TF, i);

		temp.x *= temp.w;
		temp.y *= temp.w;
		temp.z *= temp.w;

		sum.x += temp.x;
		sum.y += temp.y;
		sum.z += temp.z;
		sum.w += temp.w;
	}

	result.x = sum.x / diff; //* (newAlpha/sum.w);
	result.y = sum.y / diff; //* (newAlpha/sum.w);
	result.z = sum.z / diff; //* (newAlpha/sum.w);
	result.w = sum.w / diff;

		

	TF2d_k[TFSize*y + x].x = result.x;
	TF2d_k[TFSize*y + x].y = result.y;
	TF2d_k[TFSize*y + x].z = result.z;
	TF2d_k[TFSize*y + x].w = result.w;


}


void init_TF2d(int TFSize)
{
	int size = TFSize*TFSize;
	float4* TF2d_k;
	hipMalloc((void**)&TF2d_k, size*sizeof(float4));
	hipMemset(TF2d_k, 0, size*sizeof(float4));

	dim3 Db = dim3( 16, 16 ); 
    dim3 Dg = dim3( 256, 256 );


	TF2d_kernel<<<Dg, Db>>>(TF2d_k, TFSize); //pre-integral OTF init kernel - threads 4096*4096

	float4* TF2d;
	TF2d = new float4[size];
	memset(TF2d, 0, size*sizeof(float4));

	hipMemcpy(TF2d, TF2d_k, size*sizeof(float4), hipMemcpyDeviceToHost);

	hipFree(TF2d_k);

	initTF2dTexture(TF2d, TFSize, TFSize, 1);

	delete[] TF2d;


}

extern "C"
void GPU_Render(uchar *image, int imgsize[2], ushort* pVol, int dim[3], 
				TF *transfer, int tf_size, double zResolution, bool &bInitVol, bool &bInitTF, float *ViewingPoint)
{
	float4 *tf_cuda;
	if(!bInitTF){
		printf("-init TF texture memory - GPU\n");
		tf_cuda = new float4[tf_size];
		for(int i=0; i<tf_size; i++){
			tf_cuda[i].x = transfer[i].R;
			tf_cuda[i].y = transfer[i].G;
			tf_cuda[i].z = transfer[i].B;
			tf_cuda[i].w = transfer[i].alpha;
		}
		initTFTexture(tf_size, tf_cuda);
	}

	hipExtent volume_dim_block, volume_dim;
	float blockResolution = 0.25f;
	volume_dim = make_hipExtent(dim[0], dim[1], dim[2]);
	volume_dim_block = make_hipExtent(dim[0]*blockResolution, dim[1]*blockResolution, dim[2]*blockResolution);

	ushort *pVol_block;
	if(!bInitVol){
		printf("-init Volume texture memory - GPU\n");
		pVol_block = make_blockVolume(pVol, volume_dim_block, volume_dim);

		initVolume(pVol, volume_dim , sizeof(ushort));
		initBlockTexture(pVol_block, volume_dim_block, sizeof(ushort));
	}

	Run_Kernel(image, imgsize, volume_dim, pVol, (float)zResolution, blockResolution, ViewingPoint);

	if(!bInitVol){
		delete[] pVol_block;
		bInitVol = true;
	}
	if(!bInitTF){
		delete[] tf_cuda;
		bInitTF = true;
	}
	
}

extern "C"
void GPU_Render_AO(uchar *image, const int imgsize[2], ushort* pVol, int dim[3], 
					TF *transfer, int tf_size, double zResolution, bool &bInitVol, bool &bInitTF,
					float *Avg, float *Sig, float *Avg_half, float *Sig_half, bool &m_bInitAvgSig, 
					float probability[310], float factor[3], const float *ViewingPoint)
{
	float4 *tf_cuda;
	if(!bInitTF){
		printf("-init TF texture memory - GPU\n");
		tf_cuda = new float4[tf_size];
		for(int i=0; i<tf_size; i++){
			tf_cuda[i].x = transfer[i].R;
			tf_cuda[i].y = transfer[i].G;
			tf_cuda[i].z = transfer[i].B;
			tf_cuda[i].w = transfer[i].alpha;
		}
		initTFTexture(tf_size, tf_cuda);
	}

	hipExtent volume_dim_block, volume_dim;
	float blockResolution = 0.25f;
	volume_dim = make_hipExtent(dim[0], dim[1], dim[2]);
	volume_dim_block = make_hipExtent(dim[0]*blockResolution, dim[1]*blockResolution, dim[2]*blockResolution);

	ushort *pVol_block;
	if(!bInitVol){
		printf("-init Volume texture memory - GPU\n");
		pVol_block = make_blockVolume(pVol, volume_dim_block, volume_dim);

		initVolume(pVol, volume_dim , sizeof(ushort));
		initBlockTexture(pVol_block, volume_dim_block, sizeof(ushort));
	}
	if(!m_bInitAvgSig && Avg != NULL && Sig != NULL && Avg_half != NULL && Sig_half != NULL){
		printf("-init Avg,Sig Volume texture memory - GPU\n");
		hipExtent avg_sigma_size = make_hipExtent(dim[0]/2, dim[1]/2, dim[2]/2);

		initAvgVolume(Avg, avg_sigma_size, sizeof(float));
		initSigVolume(Sig, avg_sigma_size, sizeof(float));
		initAvgHalfVolume(Avg_half, avg_sigma_size, sizeof(float));
		initSigHalfVolume(Sig_half, avg_sigma_size, sizeof(float));
	}

	Run_Kernel_AO(image, imgsize, volume_dim, pVol, (float)zResolution, blockResolution, probability, factor, ViewingPoint);

	if(!bInitVol){
		delete[] pVol_block;
		bInitVol = true;
	}
	if(!bInitTF){
		delete[] tf_cuda;
		bInitTF = true;
	}	
}

