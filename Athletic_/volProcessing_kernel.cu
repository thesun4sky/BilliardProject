#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "TFManager.h"

#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_vector_types.h>

const float PI = 3.1415926536f;
typedef unsigned short ushort;
typedef unsigned char uchar;


__global__ void cuda_kernel_test(ushort *new_vol_k, ushort *vol_k, int3 dim3, float *gaussianMask_k, int maskSize)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	if (tx >= dim3.x-2 || ty >= dim3.y-2) return;
	if (tx <= 2 || ty <= 2) return;

	int size = maskSize/2;

	for(int tz=1; tz<dim3.z; tz++)
	{
		double sum = 0.f;
		for(int i=-size; i<size+1; i++)
		{
			for(int j=-size; j<size+1; j++)
			{
				for(int k=-size; k<size+1; k++)
				{
					int z=k+1, y=j+1, x=i+1;
					sum += vol_k[(tz+k)*dim3.x*dim3.y + (ty+j)*dim3.y + (tx+i)]*gaussianMask_k[z*maskSize*maskSize + y*maskSize + x];
				}
			}
		}
		new_vol_k[tz*dim3.x*dim3.y + ty*dim3.y + tx] = (ushort)sum;
	}

}

extern "C"
void RunSmoothFilter(ushort* pVol, int *dim)
{
	printf("-GPU RunSmoothFilter \n");

	float fSigma=0.8f;
	float gaussianMask[27];
	int maskSize = 3;
	int allocSize= maskSize*maskSize*maskSize;

	float sum=0.f;
	for(int i=0; i<maskSize; i++)
	{
		float z = fabs((float)i-1.f);
		for(int j=0; j<maskSize; j++)
		{
			float y = fabs((float)j-1.f);
			for(int k=0; k<maskSize; k++)
			{
				float x = fabs((float)k-1.f); 
				float fDist = x+y+z;
				sum += gaussianMask[k*maskSize*maskSize + j*maskSize + i] = 
					exp(-(fDist*fDist)/(2.f*fSigma*fSigma))/(sqrtf(2.f*PI)*fSigma);
			}
		}
	}
	for(int i=0; i<maskSize; i++)
	{
		for(int j=0; j<maskSize; j++)
		{
			for(int k=0; k<maskSize; k++)
			{
				gaussianMask[k*maskSize*maskSize + j*maskSize + i] /= sum;
			}
		}
	}

	//printf("%f %f %f\n", gaussianMask[0], gaussianMask[1], gaussianMask[2]);
	//printf("%f %f %f\n", gaussianMask[3], gaussianMask[4], gaussianMask[5]);
	//printf("%f %f %f\n\n", gaussianMask[6], gaussianMask[7], gaussianMask[8]);

	//printf("%f %f %f\n", gaussianMask[9], gaussianMask[10], gaussianMask[11]);
	//printf("%f %f %f\n", gaussianMask[12], gaussianMask[13], gaussianMask[14]);
	//printf("%f %f %f\n\n", gaussianMask[15], gaussianMask[16], gaussianMask[17]);

	//printf("%f %f %f\n", gaussianMask[18], gaussianMask[19], gaussianMask[20]);
	//printf("%f %f %f\n", gaussianMask[21], gaussianMask[22], gaussianMask[23]);
	//printf("%f %f %f\n\n", gaussianMask[24], gaussianMask[25], gaussianMask[26]);

	float* gaussianMask_k;
	hipMalloc((void**)&gaussianMask_k, allocSize*sizeof(float));
	hipMemset(gaussianMask_k, 0, allocSize*sizeof(float));
	hipMemcpy(gaussianMask_k, gaussianMask, allocSize*sizeof(float), hipMemcpyHostToDevice);

	ushort *pVol_k, *new_pVol_k;
	int vol_size = dim[0]*dim[1]*dim[2];
	int3 vol_dim3 = {dim[0], dim[1], dim[2]};

	hipMalloc((void**)&pVol_k, vol_size*sizeof(ushort));
	hipMemset(pVol_k, 0, vol_size*sizeof(ushort));
	hipMemcpy(pVol_k, pVol, vol_size*sizeof(ushort), hipMemcpyHostToDevice);

	hipMalloc((void**)&new_pVol_k, vol_size*sizeof(ushort));
	hipMemset(new_pVol_k, 0, vol_size*sizeof(ushort));

	dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((dim[0]+Db.x-1)/Db.x, (dim[1]+Db.y-1)/Db.y);

    cuda_kernel_test<<<Dg,Db>>>(new_pVol_k, pVol_k, vol_dim3, gaussianMask_k, maskSize);

    if (hipGetLastError() != hipSuccess)
        printf("cuda_kernel() failed to launch error = %d\n", hipGetLastError());
    
	memset(pVol, 0, sizeof(ushort)*vol_size);
	hipMemcpy(pVol, new_pVol_k, vol_size*sizeof(ushort), hipMemcpyDeviceToHost);

	hipFree(pVol_k);
	hipFree(new_pVol_k);
	hipFree(gaussianMask_k);

}


texture<ushort, 3, hipReadModeNormalizedFloat> tex_volume_processing;  // 3D texture
hipArray *d_volumeproArray = 0;


void initVolume(const ushort *h_volume, int x, int y, int z, int bytePerVoxel)
{
	hipExtent volume_Size = make_hipExtent(x, y, z);
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(bytePerVoxel*8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors( hipMalloc3DArray(&d_volumeproArray, &channelDesc, volume_Size, 0) );

    // copy data to 3D array
    hipMemcpy3DParms myParams = {0};
    myParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, x*bytePerVoxel, x, y);
    myParams.dstArray = d_volumeproArray;
    myParams.extent   = volume_Size;
    myParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&myParams) );

    // set texture parameters
    tex_volume_processing.normalized = false;                      // access with normalized texture coordinates
    tex_volume_processing.filterMode = hipFilterModeLinear;      // linear interpolation
    tex_volume_processing.channelDesc = channelDesc;
	tex_volume_processing.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex_volume_processing.addressMode[1] = hipAddressModeBorder;
    tex_volume_processing.addressMode[2] = hipAddressModeBorder;


    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_volume_processing, d_volumeproArray, channelDesc));
}


__global__ void Scaling(ushort* new_volume, float scalefactor, int fx, int fy, int fz)
{
	int tx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int ty = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	int fx_scaled = fx*scalefactor;
	int fy_scaled = fy*scalefactor;
	int fz_scaled = fz*scalefactor;

	if(tx >= fx_scaled) return;
	if(ty >= fx_scaled) return;

	float divfactor = scalefactor*scalefactor*scalefactor;

	for(int i=0; i<fz_scaled; i++)
	{
		int avgvalue=0;
		int tx_scaled = tx/scalefactor;
		int ty_scaled = ty/scalefactor;
		int tz_scaled = i/scalefactor;

		float den0 = tex3D(tex_volume_processing, tx_scaled,	ty_scaled,		tz_scaled)*65535*divfactor;
		float den1 = tex3D(tex_volume_processing, tx_scaled+1,	ty_scaled,		tz_scaled)*65535*divfactor;
		float den2 = tex3D(tex_volume_processing, tx_scaled,	ty_scaled+1,	tz_scaled)*65535*divfactor;
		float den3 = tex3D(tex_volume_processing, tx_scaled,	ty_scaled,		tz_scaled+1)*65535*divfactor;
		float den4 = tex3D(tex_volume_processing, tx_scaled+1,	ty_scaled+1,	tz_scaled)*65535*divfactor;
		float den5 = tex3D(tex_volume_processing, tx_scaled,	ty_scaled+1,	tz_scaled+1)*65535*divfactor;
		float den6 = tex3D(tex_volume_processing, tx_scaled+1,	ty_scaled,		tz_scaled+1)*65535*divfactor;
		float den7 = tex3D(tex_volume_processing, tx_scaled+1,	ty_scaled+1,	tz_scaled+1)*65535*divfactor;

		avgvalue = (int)(den0+den1+den2+den3+den4+den5+den6+den7);

		new_volume[i*fx_scaled*fy_scaled + ty*fy_scaled + tx] = avgvalue;
	}

	
}

extern "C"
ushort* RunScaling(ushort *pVol, int dim[3], float scalefactor)
{
	initVolume(pVol, dim[0], dim[1], dim[2] , sizeof(ushort));

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	//dim3 Db = dim3(32, 32);		// block dimensions are fixed to be 512 threads
    dim3 Dg = dim3((dim[0]+Db.x-1)/Db.x, (dim[1]+Db.y-1)/Db.y);
	
	int fx_scaled = dim[0]*scalefactor;
	int fy_scaled = dim[1]*scalefactor;
	int fz_scaled = dim[2]*scalefactor;
	int newsize = fx_scaled*fy_scaled*fz_scaled;

	ushort *new_vol_k;
	hipMalloc((void**)&new_vol_k, newsize*sizeof(ushort));
	hipMemset(new_vol_k, 0, newsize*sizeof(ushort));

	Scaling<<<Dg, Db>>>(new_vol_k, scalefactor, dim[0], dim[1], dim[2]);

	ushort *new_volume = new ushort[newsize];
	memset(new_volume, 0, sizeof(ushort)*newsize);

	hipMemcpy(new_volume, new_vol_k, newsize*sizeof(ushort), hipMemcpyDeviceToHost);
	
	hipFreeArray(d_volumeproArray);
	hipFree(new_vol_k);

    return new_volume;
}